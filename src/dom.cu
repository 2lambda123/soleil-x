#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <fstream>
#include <stdio.h>

#include <thrust/adjacent_difference.h>
#include <thrust/copy.h>
#include <thrust/device_delete.h>
#include <thrust/device_new.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/iterator_facade.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>

typedef std::size_t size_t;
typedef thrust::tuple<double,double> Double2;
typedef thrust::tuple<double,double,double,double> Double4;
using thrust::get;

//=============================================================================
// HELPER CLASSES
//=============================================================================

class DeviceArray;

class HostArray {
  friend class DeviceArray;
public:
  HostArray(size_t nx, size_t ny, size_t nz)
    : nx_(nx),
      ny_(ny),
      nz_(nz),
      data_(new double[nx * ny * nz]) {}
  ~HostArray() {
    delete [] data_;
  }
public:
  HostArray(const HostArray&) = delete;
  HostArray& operator=(const HostArray&) = delete;
  void copy_from(const DeviceArray& src);
  void read_from(const char* filename) {
    std::fstream fin(filename, std::fstream::in);
    for (size_t i = 0; i < nx_; ++i) {
      for (size_t j = 0; j < ny_; ++j) {
        for (size_t k = 0; k < nz_; ++k) {
          fin >> (*this)(i,j,k);
        }
      }
    }
  }
  void write_to(const char* filename) {
    FILE* fout = fopen(filename, "w");
    if (fout == NULL) {
      perror("Error opening file");
      exit(1);
    }
    for (size_t i = 0; i < nx_; ++i) {
      for (size_t j = 0; j < ny_; ++j) {
        for (size_t k = 0; k < nz_; ++k) {
          fprintf(fout, "%.17e\n", (*this)(i,j,k));
        }
      }
    }
    fclose(fout);
  }
public:
  double& operator()(size_t x, size_t y, size_t z) {
    return *(data_ + x*ny_*nz_ + y*nz_ + z);
  }
  const double& operator()(size_t x, size_t y, size_t z) const {
    return *(data_ + x*ny_*nz_ + y*nz_ + z);
  }
  size_t nx() const { return nx_; }
  size_t ny() const { return ny_; }
  size_t nz() const { return nz_; }
private:
  size_t nx_;
  size_t ny_;
  size_t nz_;
  double* data_;
};

class DeviceArray {
  friend class HostArray;
public:
  DeviceArray(size_t nx, size_t ny, size_t nz)
    : nx_(nx),
      ny_(ny),
      nz_(nz),
      data_(thrust::device_new<double>(nx * ny * nz)) {}
public:
  DeviceArray(const DeviceArray&) = delete;
  DeviceArray& operator=(const DeviceArray&) = delete;
  void copy_from(const HostArray& src) {
    assert(src.nx() == nx_ && src.ny() == ny_ && src.nz() == nz_);
    thrust::copy_n(src.data_, nx_ * ny_ * nz_, data_);
  }
public:
  thrust::device_reference<double>
  operator()(size_t x, size_t y, size_t z) {
    return *(data_ + x*ny_*nz_ + y*nz_ + z);
  }
  thrust::device_reference<const double>
  operator()(size_t x, size_t y, size_t z) const {
    return *(data_ + x*ny_*nz_ + y*nz_ + z);
  }
  thrust::device_ptr<double> z_row(size_t x, size_t y) {
    return data_ + x*ny_*nz_ + y*nz_;
  }
  thrust::device_ptr<const double> z_row(size_t x, size_t y) const {
    return data_ + x*ny_*nz_ + y*nz_;
  }
  size_t nx() const { return nx_; }
  size_t ny() const { return ny_; }
  size_t nz() const { return nz_; }
private:
  size_t nx_;
  size_t ny_;
  size_t nz_;
  thrust::device_ptr<double> data_;
};

void HostArray::copy_from(const DeviceArray& src) {
  assert(src.nx() == nx_ && src.ny() == ny_ && src.nz() == nz_);
  thrust::copy_n(src.data_, nx_ * ny_ * nz_, data_);
}

// template<typename ValueT, typename IterT> class with_header_iterator
//   : public thrust::iterator_facade<with_header_iterator<ValueT,IterT>,ValueT> {
// public:
//   typedef with_header_iterator<ValueT,IterT> SelfT;
//   typedef thrust::iterator_facade<with_header_iterator<ValueT,IterT>,
//                                   ValueT> SuperT;
//   typedef SuperT::difference_type DiffT;
//   friend class thrust::iterator_core_access;
// public:
//   __host__ __device__
//   with_header_iterator(ValueT header, IterT iter)
//     : header_(std::move(header)),
//       pos_(0),
//       iter_(std::move(iter)) {}
// private:
//   __host__ __device__
//   const ValueT& dereference() const {
//     if (pos_ == 0) {
//       return header_;
//     }
//     return *iter_;
//   }
//   __host__ __device__
//   bool equal(const SelfT& other) const {
//     return
//       // header_ == other.header_ &&
//       // iter_ == other.iter_ &&
//       pos_ == other.pos_;
//   }
//   __host__ __device__
//   void increment() {
//     if (++pos_ > 1) {
//       ++iter_;
//     }
//   }
//   __host__ __device__
//   void decrement() {
//     if (--pos_ >= 1) {
//       --iter_;
//     }
//   }
//   __host__ __device__
//   void advance(DiffT n) {
//     if (pos_ == 0) {
//       pos_++;
//       n--;
//     }
//     iter_ += n;
//   }
//   __host__ __device__
//   void distance_to(const SelfT& other) const {
//     return other.pos_ - pos_;
//   }
// private:
//   const ValueT header_;
//   DiffT pos_;
//   IterT iter_;
// };

// template<typename ValueT, typename IterT>
// add_header(ValueT header, IterT iter) {
//   return with_header_iterator(header, iter);
// }

//=============================================================================
// ALGORITHM
//=============================================================================

// Start with general TWD scheme:
// cell_int[i,j,k] = (cell_source[i,j,k] * dV
//                    + fabs(xi)  * dAx / x_gamma * x_face_int[i,j,k]
//                    + fabs(eta) * dAy / y_gamma * y_face_int[i,j,k]
//                    + fabs(mu)  * dAz / z_gamma * z_face_int[i,j,k])
//                 / (cell_sigma[i,j,k] * dV
//                    + fabs(xi)  * dAx / x_gamma
//                    + fabs(eta) * dAy / y_gamma
//                    + fabs(mu)  * dAz / z_gamma)
// x_face_int[i+1,j,k] = cell_int[i,j,k] / x_gamma
//                     - (1-x_gamma)/x_gamma * x_face_int[i,j,k]
// y_face_int[i,j+1,k] = cell_int[i,j,k] / y_gamma
//                     - (1-y_gamma)/y_gamma * y_face_int[i,j,k]
// z_face_int[i,j,k+1] = cell_int[i,j,k] / z_gamma
//                     - (1-z_gamma)/z_gamma * z_face_int[i,j,k]

// Select a specific z-row (i,j),
// treat x_face_int[i,j,*], y_face_int[i,j,*], z_face_int[i,j,0] as constants,
// reformulate in terms of z_face_int:
// a(k) = fabs(mu) * dAz / z_gamma
//      / (cell_sigma[i,j,k] * dV
//         + fabs(xi)  * dAx / x_gamma
//         + fabs(eta) * dAy / y_gamma
//         + fabs(mu)  * dAz / z_gamma)
//      / z_gamma
//      - (1-z_gamma)/z_gamma
// b(k) = (cell_source[i,j,k] * dV
//         + fabs(xi)  * dAx / x_gamma * x_face_int[i,j,k]
//         + fabs(eta) * dAy / y_gamma * y_face_int[i,j,k])
//      / (cell_sigma[i,j,k] * dV
//         + fabs(xi)  * dAx / x_gamma
//         + fabs(eta) * dAy / y_gamma
//         + fabs(mu)  * dAz / z_gamma)
//      / z_gamma
// z_face_int[i,j,k+1] = a(k) * z_face_int[i,j,k] + b(k)

// Other quantities can be computed based on z_face_int[i,j,k+1]:
// cell_int[i,j,k] = (1-z_gamma) * z_face_int[i,j,k]
//                 + z_gamma * z_face_int[i,j,k+1]
// x_face_int[i+1,j,k] = cell_int[i,j,k] / x_gamma
//                     - (1-x_gamma)/x_gamma * x_face_int[i,j,k]
// y_face_int[i,j+1,k] = cell_int[i,j,k] / y_gamma
//                     - (1-y_gamma)/y_gamma * y_face_int[i,j,k]

// To solve the recurrence relation for z_face_int:
// Create a vector of pairs:
// (any,z_face_int[i,j,0]) (a(0),b(0)) ... (a(NZ-1),b(NZ-1))
// Do a prefix sum using operator:
// (a,b) X (c,d) = ( a*c, (b*c)+d )
// Take the 2nd element of each pair.

struct A : public thrust::unary_function<double,double> {
  __host__ __device__
  double operator()(double sigma) const {
    double dAx = dy_*dz_;
    double dAy = dx_*dz_;
    double dAz = dx_*dy_;
    double dV = dx_*dy_*dz_;
    return
      fabs(mu_) * dAz / z_gamma_
      / (sigma * dV
         + fabs(xi_)  * dAx / x_gamma_
         + fabs(eta_) * dAy / y_gamma_
         + fabs(mu_)  * dAz / z_gamma_)
      / z_gamma_
      - (1-z_gamma_)/z_gamma_;
  }
  __host__ __device__
  A(double dx, double dy, double dz,
    double xi, double eta, double mu,
    double x_gamma, double y_gamma, double z_gamma)
    : dx_(dx), dy_(dy), dz_(dz),
      xi_(xi), eta_(eta), mu_(mu),
      x_gamma_(x_gamma), y_gamma_(y_gamma), z_gamma_(z_gamma) {}
  double dx_, dy_, dz_;
  double xi_, eta_, mu_;
  double x_gamma_, y_gamma_, z_gamma_;
};

struct B : public thrust::unary_function<Double4,double> {
  __host__ __device__
  double operator()(const Double4& args) const {
    double cell_source = get<0>(args);
    double x_face_int = get<1>(args);
    double y_face_int = get<2>(args);
    double cell_sigma = get<3>(args);
    double dAx = dy_*dz_;
    double dAy = dx_*dz_;
    double dAz = dx_*dy_;
    double dV = dx_*dy_*dz_;
    return
      (cell_source * dV
       + fabs(xi_)  * dAx / x_gamma_ * x_face_int
       + fabs(eta_) * dAy / y_gamma_ * y_face_int)
      / (cell_sigma * dV
         + fabs(xi_)  * dAx / x_gamma_
         + fabs(eta_) * dAy / y_gamma_
         + fabs(mu_)  * dAz / z_gamma_)
      / z_gamma_;
  }
  __host__ __device__
  B(double dx, double dy, double dz,
    double xi, double eta, double mu,
    double x_gamma, double y_gamma, double z_gamma)
    : dx_(dx), dy_(dy), dz_(dz),
      xi_(xi), eta_(eta), mu_(mu),
      x_gamma_(x_gamma), y_gamma_(y_gamma), z_gamma_(z_gamma) {}
  double dx_, dy_, dz_;
  double xi_, eta_, mu_;
  double x_gamma_, y_gamma_, z_gamma_;
};

struct X : public thrust::binary_function<Double2,Double2,Double2> {
  __host__ __device__
  Double2 operator()(const Double2& lhs, const Double2& rhs) const {
    return thrust::make_tuple(get<0>(lhs) * get<0>(rhs),
                              get<1>(lhs) * get<0>(rhs) + get<1>(rhs));
  }
};

struct FaceInt2CellInt : public thrust::unary_function<Double2,double> {
  __host__ __device__
  double operator()(const Double2& args) const {
    double face_int_prev = get<0>(args);
    double face_int_next = get<1>(args);
    return (1-gamma_) * face_int_prev + gamma_ * face_int_next;
  }
  __host__ __device__
  FaceInt2CellInt(double gamma) : gamma_(gamma) {}
  double gamma_;
};

struct CellInt2FaceInt : public thrust::unary_function<Double2,double> {
  __host__ __device__
  double operator()(const Double2& args) const {
    double cell_int = get<0>(args);
    double face_int_prev = get<1>(args);
    return cell_int / gamma_ - (1-gamma_)/gamma_ * face_int_prev;
  }
  __host__ __device__
  CellInt2FaceInt(double gamma) : gamma_(gamma) {}
  double gamma_;
};

struct Project2nd : public thrust::unary_function<Double2,double> {
  __host__ __device__
  double operator()(const Double2& tup) const {
    return get<1>(tup);
  }
};

void sweep(DeviceArray/*(NX,NY,NZ)*/& cell_int,
           const DeviceArray/*(NX,NY,NZ)*/& cell_source,
           const DeviceArray/*(NX,NY,NZ)*/& cell_sigma,
           DeviceArray/*(NX+1,NY,NZ)*/& x_face_int,
           DeviceArray/*(NX,NY+1,NZ)*/& y_face_int,
           DeviceArray/*(NX,NY,NZ+1)*/& z_face_int,
           double dx, double dy, double dz,
           double xi, double eta, double mu,
           double x_gamma, double y_gamma, double z_gamma) {
  size_t NX = cell_int.nx();
  size_t NY = cell_int.ny();
  size_t NZ = cell_int.nz();
  // Construct function objects
  A a(dx, dy, dz, xi, eta, mu, x_gamma, y_gamma, z_gamma);
  B b(dx, dy, dz, xi, eta, mu, x_gamma, y_gamma, z_gamma);
  X x;
  FaceInt2CellInt z2c(z_gamma);
  CellInt2FaceInt c2x(x_gamma);
  CellInt2FaceInt c2y(y_gamma);
  // Iterate over all z-rows, in row-major order
  for (size_t i = 0; i < NX; ++i) {
    for (size_t j = 0; j < NY; ++j) {
      // Fill in pairs
      thrust::device_vector<Double2> pairs(NZ+1);
      pairs[0] = thrust::make_tuple(-1.0, z_face_int(i,j,0));
      thrust::copy_n(
        thrust::make_zip_iterator(thrust::make_tuple(
          thrust::make_transform_iterator(
            cell_sigma.z_row(i,j),
            a),
          thrust::make_transform_iterator(
            thrust::make_zip_iterator(thrust::make_tuple(
              cell_source.z_row(i,j),
              x_face_int.z_row(i,j),
              y_face_int.z_row(i,j),
              cell_sigma.z_row(i,j))),
            b))),
        NZ,
        pairs.begin() + 1);
      // Run prefix sum over pairs
      thrust::inclusive_scan(pairs.begin(), pairs.end(), pairs.begin(), x);
      // Retrieve z-face intensity values
      thrust::copy_n(
        thrust::make_transform_iterator(
          pairs.begin() + 1,
          Project2nd()),
        NZ,
        z_face_int.z_row(i,j) + 1);
      // Calculate cell intensity values
      thrust::copy_n(
        thrust::make_transform_iterator(
          thrust::make_zip_iterator(thrust::make_tuple(
            z_face_int.z_row(i,j),
            z_face_int.z_row(i,j) + 1)),
          z2c),
        NZ,
        cell_int.z_row(i,j));
      // Calculate downstream x-face intensity values
      thrust::copy_n(
        thrust::make_transform_iterator(
          thrust::make_zip_iterator(thrust::make_tuple(
            cell_int.z_row(i,j),
            x_face_int.z_row(i,j))),
          c2x),
        NZ,
        x_face_int.z_row(i+1,j));
      // Calculate downstream y-face intensity values
      thrust::copy_n(
        thrust::make_transform_iterator(
          thrust::make_zip_iterator(thrust::make_tuple(
            cell_int.z_row(i,j),
            y_face_int.z_row(i,j))),
          c2y),
        NZ,
        y_face_int.z_row(i,j+1));
    }
  }
}

//=============================================================================
// MAIN
//=============================================================================

int main() {
  size_t NX, NY, NZ;
  double x_gamma, y_gamma, z_gamma;
  double dx, dy, dz;
  double xi, eta, mu;

  {
    std::fstream setup("setup.dat", std::fstream::in);
    setup >> NX >> NY >> NZ;
    setup >> x_gamma >> y_gamma >> z_gamma;
    setup >> dx >> dy >> dz;
    setup >> xi >> eta >> mu;
  }

  HostArray h_cell_int(NX,NY,NZ);
  HostArray h_cell_source(NX,NY,NZ);
  HostArray h_cell_sigma(NX,NY,NZ);
  HostArray h_x_face_int(NX+1,NY,NZ);
  HostArray h_y_face_int(NX,NY+1,NZ);
  HostArray h_z_face_int(NX,NY,NZ+1);

  DeviceArray d_cell_int(NX,NY,NZ);
  DeviceArray d_cell_source(NX,NY,NZ);
  DeviceArray d_cell_sigma(NX,NY,NZ);
  DeviceArray d_x_face_int(NX+1,NY,NZ);
  DeviceArray d_y_face_int(NX,NY+1,NZ);
  DeviceArray d_z_face_int(NX,NY,NZ+1);

  h_cell_int.read_from("cell_int_prev.dat");
  h_cell_source.read_from("cell_source.dat");
  h_cell_sigma.read_from("cell_sigma.dat");
  h_x_face_int.read_from("x_face_int_prev.dat");
  h_y_face_int.read_from("y_face_int_prev.dat");
  h_z_face_int.read_from("z_face_int_prev.dat");

  d_cell_int.copy_from(h_cell_int);
  d_cell_source.copy_from(h_cell_source);
  d_cell_sigma.copy_from(h_cell_sigma);
  d_x_face_int.copy_from(h_x_face_int);
  d_y_face_int.copy_from(h_y_face_int);
  d_z_face_int.copy_from(h_z_face_int);

  sweep(d_cell_int,
        d_cell_source,
        d_cell_sigma,
        d_x_face_int,
        d_y_face_int,
        d_z_face_int,
        dx, dy, dz,
        xi, eta, mu,
        x_gamma, y_gamma, z_gamma);

  h_cell_int.copy_from(d_cell_int);
  h_x_face_int.copy_from(d_x_face_int);
  h_y_face_int.copy_from(d_y_face_int);
  h_z_face_int.copy_from(d_z_face_int);

  h_cell_int.write_to("cell_int_cu.dat");
  h_x_face_int.write_to("x_face_int_cu.dat");
  h_y_face_int.write_to("y_face_int_cu.dat");
  h_z_face_int.write_to("z_face_int_cu.dat");
}
